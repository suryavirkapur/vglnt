#include "hip/hip_runtime.h"
#include "lstm_cuda_kernels.cuh"
#include <hip/hip_runtime.h>

namespace lstm
{
    namespace cuda
    {

        __global__ void matrix_multiply_kernel(const float *A, const float *B, float *C,
                                               int M, int N, int K)
        {
            int row = blockIdx.y * blockDim.y + threadIdx.y;
            int col = blockIdx.x * blockDim.x + threadIdx.x;

            if (row < M && col < N)
            {
                float sum = 0.0f;
                for (int i = 0; i < K; ++i)
                {
                    sum += A[row * K + i] * B[i * N + col];
                }
                C[row * N + col] = sum;
            }
        }

        __global__ void sigmoid_kernel(float *data, int size)
        {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if (idx < size)
            {
                data[idx] = 1.0f / (1.0f + expf(-data[idx]));
            }
        }

        __global__ void tanh_kernel(float *data, int size)
        {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if (idx < size)
            {
                data[idx] = tanhf(data[idx]);
            }
        }

        void matrix_multiply(const float *A, const float *B, float *C,
                             int M, int N, int K, hipStream_t stream)
        {
            dim3 block(16, 16);
            dim3 grid((N + block.x - 1) / block.x,
                      (M + block.y - 1) / block.y);

            matrix_multiply_kernel<<<grid, block, 0, stream>>>(A, B, C, M, N, K);
        }

        void sigmoid_activation(float *data, int size, hipStream_t stream)
        {
            int block_size = 256;
            int num_blocks = (size + block_size - 1) / block_size;
            sigmoid_kernel<<<num_blocks, block_size, 0, stream>>>(data, size);
        }

        void tanh_activation(float *data, int size, hipStream_t stream)
        {
            int block_size = 256;
            int num_blocks = (size + block_size - 1) / block_size;
            tanh_kernel<<<num_blocks, block_size, 0, stream>>>(data, size);
        }

    }
}